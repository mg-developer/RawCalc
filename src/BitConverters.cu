#include "hip/hip_runtime.h"
#include "RawStructs.h"
#include "ManagedMem.h"
#include <memory>

managed_data_block to16(datatype rData)
{
    // preparing variables
    int resx = rData.metaData.xResolution;
    int resy = rData.metaData.yResolution;
    int bl = rData.metaData.blackLevelOld;
    bool maximize = rData.metaData.maximize;
    double maximizer = rData.metaData.maximizer;

    unsigned char* source = rData.rawData;
    // ------------- and go ----
    unsigned int chunks = resx * resy * 14 / 8;
    managed_data_block Dest(std::size_t(chunks / 14 * 16));
   // unique_data_block Dest(new unsigned char[chunks / 14 * 16]);
   
    Dest[3] =3;
    unsigned int tt = 0;
    int senselA, senselB, senselC, senselD, senselE, senselF, senselG, senselH;
    for (unsigned int t = 0; t < chunks; t += 14)
    {
        if (maximize == true)
        {
            senselA = (int)((source[t] >> 2) | (source[t + 1] << 6));
            senselB = (int)(((source[t] & 0x3) << 12) | (source[t + 3] << 4) | (source[t + 2] >> 4));
            senselC = (int)(((source[t + 2] & 0x0f) << 10) | (source[t + 5] << 2) | (source[t + 4] >> 6));
            senselD = (int)(((source[t + 4] & 0x3f) << 8) | (source[t + 7]));
            senselE = (int)((source[t + 9] >> 2) | (source[t + 6] << 6));
            senselF = (int)(((source[t + 9] & 0x3) << 12) | (source[t + 8] << 4) | (source[t + 11] >> 4));
            senselG = (int)(((source[t + 11] & 0x0f) << 10) | (source[t + 10] << 2) | (source[t + 13] >> 6));
            senselH = (int)(((source[t + 13] & 0x3f) << 8) | (source[t + 12]));

            // debias sensel
            senselA = senselA - (int)bl;
            senselB = senselB - (int)bl;
            senselC = senselC - (int)bl;
            senselD = senselD - (int)bl;
            senselE = senselE - (int)bl;
            senselF = senselF - (int)bl;
            senselG = senselG - (int)bl;
            senselH = senselH - (int)bl;

            // maximize to 16bit
            senselA = (int)(senselA * maximizer);
            senselB = (int)(senselB * maximizer);
            senselC = (int)(senselC * maximizer);
            senselD = (int)(senselD * maximizer);
            senselE = (int)(senselE * maximizer);
            senselF = (int)(senselF * maximizer);
            senselG = (int)(senselG * maximizer);
            senselH = (int)(senselH * maximizer);

            // do max on overflow
            if (senselA > 65535) senselA = 65535;
            if (senselB > 65535) senselB = 65535;
            if (senselC > 65535) senselC = 65535;
            if (senselD > 65535) senselD = 65535;
            if (senselE > 65535) senselE = 65535;
            if (senselF > 65535) senselF = 65535;
            if (senselG > 65535) senselG = 65535;
            if (senselH > 65535) senselH = 65535;

            // -- react on underflow
            if (senselA < 0) senselA = 0;
            if (senselB < 0) senselB = 0;
            if (senselC < 0) senselC = 0;
            if (senselD < 0) senselD = 0;
            if (senselE < 0) senselE = 0;
            if (senselF < 0) senselF = 0;
            if (senselG < 0) senselG = 0;
            if (senselH < 0) senselH = 0;

        }
        else
        {
            // no maximizing
            senselA = (int)((source[t] >> 2) | (source[t + 1] << 6));
            senselB = (int)(((source[t] & 0x3) << 12) | (source[t + 3] << 4) | (source[t + 2] >> 4));
            senselC = (int)(((source[t + 2] & 0x0f) << 10) | (source[t + 5] << 2) | (source[t + 4] >> 6));
            senselD = (int)(((source[t + 4] & 0x3f) << 8) | (source[t + 7]));
            senselE = (int)((source[t + 9] >> 2) | (source[t + 6] << 6));
            senselF = (int)(((source[t + 9] & 0x3) << 12) | (source[t + 8] << 4) | (source[t + 11] >> 4));
            senselG = (int)(((source[t + 11] & 0x0f) << 10) | (source[t + 10] << 2) | (source[t + 13] >> 6));
            senselH = (int)(((source[t + 13] & 0x3f) << 8) | (source[t + 12]));

        }

        Dest[tt++] = (unsigned char)(senselA & 0xff);
        Dest[tt++] = (unsigned char)(senselA >> 8);

        Dest[tt++] = (unsigned char)(senselB & 0xff);
        Dest[tt++] = (unsigned char)(senselB >> 8);

        Dest[tt++] = (unsigned char)(senselC & 0xff);
        Dest[tt++] = (unsigned char)(senselC >> 8);

        Dest[tt++] = (unsigned char)(senselD & 0xff);
        Dest[tt++] = (unsigned char)(senselD >> 8);

        Dest[tt++] = (unsigned char)(senselE & 0xff);
        Dest[tt++] = (unsigned char)(senselE >> 8);

        Dest[tt++] = (unsigned char)(senselF & 0xff);
        Dest[tt++] = (unsigned char)(senselF >> 8);

        Dest[tt++] = (unsigned char)(senselG & 0xff);
        Dest[tt++] = (unsigned char)(senselG >> 8);

        Dest[tt++] = (unsigned char)(senselH & 0xff);
        Dest[tt++] = (unsigned char)(senselH >> 8);

    }
    return std::move(Dest);
}

// 5DIII valuerange
// 14bit original - 2048-15.000 = ~12.900
// 16bit - 8192-60.000 - maximized 0-65535
// 12bit - 512-3750 = ~3.200 - maximized 0-4095 (

managed_data_block from16to12(datatype rData)
{
    // preparing variables
    int resx = rData.metaData.xResolution;
    int resy = rData.metaData.yResolution;
    unsigned char* source = rData.rawData;
    // ------------- and go ----

    unsigned int chunks = resx * resy * 16 / 8;
    managed_data_block Dest(std::size_t(chunks / 16 * 12 + 72));
    unsigned int tt = 0;
    int senselA, senselB, senselC, senselD, senselE, senselF, senselG, senselH;
    int senselI, senselJ, senselK, senselL, senselM, senselN, senselO, senselP;
    int senselQ, senselR, senselS, senselT, senselU, senselV, senselW, senselX;

    for (unsigned int t = 0; t < chunks; t += 48)
    {
        //read 16bit data and shift 4 bits.
        senselA = (source[t] | (source[t + 1] << 8)) >> 4;
        senselB = (source[t + 2] | (source[t + 3] << 8)) >> 4;
        senselC = (source[t + 4] | (source[t + 5] << 8)) >> 4;
        senselD = (source[t + 6] | (source[t + 7] << 8)) >> 4;
        senselE = (source[t + 8] | (source[t + 9] << 8)) >> 4;
        senselF = (source[t + 10] | (source[t + 11] << 8)) >> 4;
        senselG = (source[t + 12] | (source[t + 13] << 8)) >> 4;
        senselH = (source[t + 14] | (source[t + 15] << 8)) >> 4;

        senselI = (source[t + 16] | (source[t + 17] << 8)) >> 4;
        senselJ = (source[t + 18] | (source[t + 19] << 8)) >> 4;
        senselK = (source[t + 20] | (source[t + 21] << 8)) >> 4;
        senselL = (source[t + 22] | (source[t + 23] << 8)) >> 4;
        senselM = (source[t + 24] | (source[t + 25] << 8)) >> 4;
        senselN = (source[t + 26] | (source[t + 27] << 8)) >> 4;
        senselO = (source[t + 28] | (source[t + 29] << 8)) >> 4;
        senselP = (source[t + 30] | (source[t + 31] << 8)) >> 4;

        senselQ = (source[t + 32] | (source[t + 33] << 8)) >> 4;
        senselR = (source[t + 34] | (source[t + 35] << 8)) >> 4;
        senselS = (source[t + 36] | (source[t + 37] << 8)) >> 4;
        senselT = (source[t + 38] | (source[t + 39] << 8)) >> 4;
        senselU = (source[t + 40] | (source[t + 41] << 8)) >> 4;
        senselV = (source[t + 42] | (source[t + 43] << 8)) >> 4;
        senselW = (source[t + 44] | (source[t + 45] << 8)) >> 4;
        senselX = (source[t + 46] | (source[t + 47] << 8)) >> 4;

        Dest[tt++] = (unsigned char)((senselA >> 4) & 0xff);
        Dest[tt++] = (unsigned char)(((senselA & 0xF) << 4) | (senselB >> 8));
        Dest[tt++] = (unsigned char)(senselB & 0xff);

        Dest[tt++] = (unsigned char)((senselC >> 4) & 0xff);
        Dest[tt++] = (unsigned char)(((senselC & 0xF) << 4) | (senselD >> 8));
        Dest[tt++] = (unsigned char)(senselD & 0xff);

        Dest[tt++] = (unsigned char)((senselE >> 4) & 0xff);
        Dest[tt++] = (unsigned char)(((senselE & 0xF) << 4) | (senselF >> 8));
        Dest[tt++] = (unsigned char)(senselF & 0xff);
        //9
        Dest[tt++] = (unsigned char)((senselG >> 4) & 0xff);
        Dest[tt++] = (unsigned char)(((senselG & 0xF) << 4) | (senselH >> 8));
        Dest[tt++] = (unsigned char)(senselH & 0xff);

        Dest[tt++] = (unsigned char)((senselI >> 4) & 0xff);
        Dest[tt++] = (unsigned char)(((senselI & 0xF) << 4) | (senselJ >> 8));
        Dest[tt++] = (unsigned char)(senselJ & 0xff);

        Dest[tt++] = (unsigned char)((senselK >> 4) & 0xff);
        Dest[tt++] = (unsigned char)(((senselK & 0xF) << 4) | (senselL >> 8));
        Dest[tt++] = (unsigned char)(senselL & 0xff);
        //18
        Dest[tt++] = (unsigned char)((senselM >> 4) & 0xff);
        Dest[tt++] = (unsigned char)(((senselM & 0xF) << 4) | (senselN >> 8));
        Dest[tt++] = (unsigned char)(senselN & 0xff);

        Dest[tt++] = (unsigned char)((senselO >> 4) & 0xff);
        Dest[tt++] = (unsigned char)(((senselO & 0xF) << 4) | (senselP >> 8));
        Dest[tt++] = (unsigned char)(senselP & 0xff);

        Dest[tt++] = (unsigned char)((senselQ >> 4) & 0xff);
        Dest[tt++] = (unsigned char)(((senselQ & 0xF) << 4) | (senselR >> 8));
        Dest[tt++] = (unsigned char)(senselR & 0xff);
        //27
        Dest[tt++] = (unsigned char)((senselS >> 4) & 0xff);
        Dest[tt++] = (unsigned char)(((senselS & 0xF) << 4) | (senselT >> 8));
        Dest[tt++] = (unsigned char)(senselT & 0xff);

        Dest[tt++] = (unsigned char)((senselU >> 4) & 0xff);
        Dest[tt++] = (unsigned char)(((senselU & 0xF) << 4) | (senselV >> 8));
        Dest[tt++] = (unsigned char)(senselV & 0xff);

        Dest[tt++] = (unsigned char)((senselW >> 4) & 0xff);
        Dest[tt++] = (unsigned char)(((senselW & 0xF) << 4) | (senselX >> 8));
        Dest[tt++] = (unsigned char)(senselX & 0xff);
        //36
    }
    return std::move(Dest);
}

managed_data_block to12(datatype rData)
{
    // preparing variables
    int resx = rData.metaData.xResolution;
    int resy = rData.metaData.yResolution;
    int bl = rData.metaData.blackLevelOld;
    bool maximize = rData.metaData.maximize;
    double maximizer = rData.metaData.maximizer;
    unsigned char* source = rData.rawData;
    // ------------- and go ----

    unsigned int chunks = resx * resy * 14 / 8;
    managed_data_block Dest(std::size_t(chunks / 14 * 12 + 42));
    unsigned int tt = 0;
    int senselA, senselB, senselC, senselD, senselE, senselF, senselG, senselH;
    int senselI, senselJ, senselK, senselL, senselM, senselN, senselO, senselP;
    int senselQ, senselR, senselS, senselT, senselU, senselV, senselW, senselX;

    for (unsigned int t = 0; t < chunks; t += 42)
    {
        if (maximize == true)
        {
            senselA = (int)((source[t] >> 2) | (source[t + 1] << 6)) - (int)bl;
            senselB = (int)(((source[t] & 0x3) << 12) | (source[t + 3] << 4) | (source[t + 2] >> 4)) - (int)bl;
            senselC = (int)(((source[t + 2] & 0x0f) << 10) | (source[t + 5] << 2) | (source[t + 4] >> 6)) - (int)bl;
            senselD = (int)(((source[t + 4] & 0x3f) << 8) | (source[t + 7])) - (int)bl;
            senselE = (int)((source[t + 9] >> 2) | (source[t + 6] << 6)) - (int)bl;
            senselF = (int)(((source[t + 9] & 0x3) << 12) | (source[t + 8] << 4) | (source[t + 11] >> 4)) - (int)bl;
            senselG = (int)(((source[t + 11] & 0x0f) << 10) | (source[t + 10] << 2) | (source[t + 13] >> 6)) - (int)bl;
            senselH = (int)(((source[t + 13] & 0x3f) << 8) | (source[t + 12])) - (int)bl;

            senselI = (int)((source[t + 14] >> 2) | (source[t + 15] << 6)) - (int)bl;
            senselJ = (int)(((source[t + 14] & 0x3) << 12) | (source[t + 17] << 4) | (source[t + 16] >> 4)) - (int)bl;
            senselK = (int)(((source[t + 16] & 0x0f) << 10) | (source[t + 19] << 2) | (source[t + 18] >> 6)) - (int)bl;
            senselL = (int)(((source[t + 18] & 0x3f) << 8) | (source[t + 21])) - (int)bl;
            senselM = (int)((source[t + 23] >> 2) | (source[t + 20] << 6)) - (int)bl;
            senselN = (int)(((source[t + 23] & 0x3) << 12) | (source[t + 22] << 4) | (source[t + 25] >> 4)) - (int)bl;
            senselO = (int)(((source[t + 25] & 0x0f) << 10) | (source[t + 24] << 2) | (source[t + 27] >> 6)) - (int)bl;
            senselP = (int)(((source[t + 27] & 0x3f) << 8) | (source[t + 26])) - (int)bl;

            senselQ = (int)((source[t + 28] >> 2) | (source[t + 29] << 6)) - (int)bl;
            senselR = (int)(((source[t + 28] & 0x3) << 12) | (source[t + 31] << 4) | (source[t + 30] >> 4)) - (int)bl;
            senselS = (int)(((source[t + 30] & 0x0f) << 10) | (source[t + 33] << 2) | (source[t + 32] >> 6)) - (int)bl;
            senselT = (int)(((source[t + 32] & 0x3f) << 8) | (source[t + 35])) - (int)bl;
            senselU = (int)((source[t + 37] >> 2) | (source[t + 34] << 6)) - (int)bl;
            senselV = (int)(((source[t + 37] & 0x3) << 12) | (source[t + 36] << 4) | (source[t + 39] >> 4)) - (int)bl;
            senselW = (int)(((source[t + 39] & 0x0f) << 10) | (source[t + 38] << 2) | (source[t + 41] >> 6)) - (int)bl;
            senselX = (int)(((source[t + 41] & 0x3f) << 8) | (source[t + 40])) - (int)bl;

            // maximize to 12bit
            senselA = (int)(senselA * maximizer);
            senselB = (int)(senselB * maximizer);
            senselC = (int)(senselC * maximizer);
            senselD = (int)(senselD * maximizer);
            senselE = (int)(senselE * maximizer);
            senselF = (int)(senselF * maximizer);
            senselG = (int)(senselG * maximizer);
            senselH = (int)(senselH * maximizer);
            senselI = (int)(senselI * maximizer);
            senselJ = (int)(senselJ * maximizer);
            senselK = (int)(senselK * maximizer);
            senselL = (int)(senselL * maximizer);
            senselM = (int)(senselM * maximizer);
            senselN = (int)(senselN * maximizer);
            senselO = (int)(senselO * maximizer);
            senselP = (int)(senselP * maximizer);
            senselQ = (int)(senselQ * maximizer);
            senselR = (int)(senselR * maximizer);
            senselS = (int)(senselS * maximizer);
            senselT = (int)(senselT * maximizer);
            senselU = (int)(senselU * maximizer);
            senselV = (int)(senselV * maximizer);
            senselW = (int)(senselW * maximizer);
            senselX = (int)(senselX * maximizer);

            // check on overflow
            if (senselA > 4095) senselA = 4095;
            if (senselB > 4095) senselB = 4095;
            if (senselC > 4095) senselC = 4095;
            if (senselD > 4095) senselD = 4095;
            if (senselE > 4095) senselE = 4095;
            if (senselF > 4095) senselF = 4095;
            if (senselG > 4095) senselG = 4095;
            if (senselH > 4095) senselH = 4095;
            if (senselI > 4095) senselI = 4095;
            if (senselJ > 4095) senselJ = 4095;
            if (senselK > 4095) senselK = 4095;
            if (senselL > 4095) senselL = 4095;
            if (senselM > 4095) senselM = 4095;
            if (senselN > 4095) senselN = 4095;
            if (senselO > 4095) senselO = 4095;
            if (senselP > 4095) senselP = 4095;
            if (senselQ > 4095) senselQ = 4095;
            if (senselR > 4095) senselR = 4095;
            if (senselS > 4095) senselS = 4095;
            if (senselT > 4095) senselT = 4095;
            if (senselU > 4095) senselU = 4095;
            if (senselV > 4095) senselV = 4095;
            if (senselW > 4095) senselW = 4095;
            if (senselX > 4095) senselX = 4095;


            // -- react on underflow
            if (senselA < 0) senselA = 0;
            if (senselB < 0) senselB = 0;
            if (senselC < 0) senselC = 0;
            if (senselD < 0) senselD = 0;
            if (senselE < 0) senselE = 0;
            if (senselF < 0) senselF = 0;
            if (senselG < 0) senselG = 0;
            if (senselH < 0) senselH = 0;
            if (senselI < 0) senselI = 0;
            if (senselJ < 0) senselJ = 0;
            if (senselK < 0) senselK = 0;
            if (senselL < 0) senselL = 0;
            if (senselM < 0) senselM = 0;
            if (senselN < 0) senselN = 0;
            if (senselO < 0) senselO = 0;
            if (senselP < 0) senselP = 0;
            if (senselQ < 0) senselQ = 0;
            if (senselR < 0) senselR = 0;
            if (senselS < 0) senselS = 0;
            if (senselT < 0) senselT = 0;
            if (senselU < 0) senselU = 0;
            if (senselV < 0) senselV = 0;
            if (senselW < 0) senselW = 0;
            if (senselX < 0) senselX = 0;

        }
        else
        {
            senselA = (int)((source[t] >> 2) | (source[t + 1] << 6));
            senselB = (int)(((source[t] & 0x3) << 12) | (source[t + 3] << 4) | (source[t + 2] >> 4));
            senselC = (int)(((source[t + 2] & 0x0f) << 10) | (source[t + 5] << 2) | (source[t + 4] >> 6));
            senselD = (int)(((source[t + 4] & 0x3f) << 8) | (source[t + 7]));
            senselE = (int)((source[t + 9] >> 2) | (source[t + 6] << 6));
            senselF = (int)(((source[t + 9] & 0x3) << 12) | (source[t + 8] << 4) | (source[t + 11] >> 4));
            senselG = (int)(((source[t + 11] & 0x0f) << 10) | (source[t + 10] << 2) | (source[t + 13] >> 6));
            senselH = (int)(((source[t + 13] & 0x3f) << 8) | (source[t + 12]));

            senselI = (int)((source[t + 14] >> 2) | (source[t + 15] << 6));
            senselJ = (int)(((source[t + 14] & 0x3) << 12) | (source[t + 17] << 4) | (source[t + 16] >> 4));
            senselK = (int)(((source[t + 16] & 0x0f) << 10) | (source[t + 19] << 2) | (source[t + 18] >> 6));
            senselL = (int)(((source[t + 18] & 0x3f) << 8) | (source[t + 21]));
            senselM = (int)((source[t + 23] >> 2) | (source[t + 20] << 6));
            senselN = (int)(((source[t + 23] & 0x3) << 12) | (source[t + 22] << 4) | (source[t + 25] >> 4));
            senselO = (int)(((source[t + 25] & 0x0f) << 10) | (source[t + 24] << 2) | (source[t + 27] >> 6));
            senselP = (int)(((source[t + 27] & 0x3f) << 8) | (source[t + 26]));

            senselQ = (int)((source[t + 28] >> 2) | (source[t + 29] << 6));
            senselR = (int)(((source[t + 28] & 0x3) << 12) | (source[t + 31] << 4) | (source[t + 30] >> 4));
            senselS = (int)(((source[t + 30] & 0x0f) << 10) | (source[t + 33] << 2) | (source[t + 32] >> 6));
            senselT = (int)(((source[t + 32] & 0x3f) << 8) | (source[t + 35]));
            senselU = (int)((source[t + 37] >> 2) | (source[t + 34] << 6));
            senselV = (int)(((source[t + 37] & 0x3) << 12) | (source[t + 36] << 4) | (source[t + 39] >> 4));
            senselW = (int)(((source[t + 39] & 0x0f) << 10) | (source[t + 38] << 2) | (source[t + 41] >> 6));
            senselX = (int)(((source[t + 41] & 0x3f) << 8) | (source[t + 40]));
            senselA = senselA >> 2;
            senselB = senselB >> 2;
            senselC = senselC >> 2;
            senselD = senselD >> 2;
            senselE = senselE >> 2;
            senselF = senselF >> 2;
            senselG = senselG >> 2;
            senselH = senselH >> 2;
            senselI = senselI >> 2;
            senselJ = senselJ >> 2;
            senselK = senselK >> 2;
            senselL = senselL >> 2;
            senselM = senselM >> 2;
            senselN = senselN >> 2;
            senselO = senselO >> 2;
            senselP = senselP >> 2;
            senselQ = senselQ >> 2;
            senselR = senselR >> 2;
            senselS = senselS >> 2;
            senselT = senselT >> 2;
            senselU = senselU >> 2;
            senselV = senselV >> 2;
            senselW = senselW >> 2;
            senselX = senselX >> 2;
        }

        Dest[tt++] = (unsigned char)((senselA >> 4) & 0xff);
        Dest[tt++] = (unsigned char)(((senselA & 0xF) << 4) | (senselB >> 8));
        Dest[tt++] = (unsigned char)(senselB & 0xff);

        Dest[tt++] = (unsigned char)((senselC >> 4) & 0xff);
        Dest[tt++] = (unsigned char)(((senselC & 0xF) << 4) | (senselD >> 8));
        Dest[tt++] = (unsigned char)(senselD & 0xff);

        Dest[tt++] = (unsigned char)((senselE >> 4) & 0xff);
        Dest[tt++] = (unsigned char)(((senselE & 0xF) << 4) | (senselF >> 8));
        Dest[tt++] = (unsigned char)(senselF & 0xff);

        Dest[tt++] = (unsigned char)((senselG >> 4) & 0xff);
        Dest[tt++] = (unsigned char)(((senselG & 0xF) << 4) | (senselH >> 8));
        Dest[tt++] = (unsigned char)(senselH & 0xff);

        Dest[tt++] = (unsigned char)((senselI >> 4) & 0xff);
        Dest[tt++] = (unsigned char)(((senselI & 0xF) << 4) | (senselJ >> 8));
        Dest[tt++] = (unsigned char)(senselJ & 0xff);

        Dest[tt++] = (unsigned char)((senselK >> 4) & 0xff);
        Dest[tt++] = (unsigned char)(((senselK & 0xF) << 4) | (senselL >> 8));
        Dest[tt++] = (unsigned char)(senselL & 0xff);

        Dest[tt++] = (unsigned char)((senselM >> 4) & 0xff);
        Dest[tt++] = (unsigned char)(((senselM & 0xF) << 4) | (senselN >> 8));
        Dest[tt++] = (unsigned char)(senselN & 0xff);

        Dest[tt++] = (unsigned char)((senselO >> 4) & 0xff);
        Dest[tt++] = (unsigned char)(((senselO & 0xF) << 4) | (senselP >> 8));
        Dest[tt++] = (unsigned char)(senselP & 0xff);

        Dest[tt++] = (unsigned char)((senselQ >> 4) & 0xff);
        Dest[tt++] = (unsigned char)(((senselQ & 0xF) << 4) | (senselR >> 8));
        Dest[tt++] = (unsigned char)(senselR & 0xff);

        Dest[tt++] = (unsigned char)((senselS >> 4) & 0xff);
        Dest[tt++] = (unsigned char)(((senselS & 0xF) << 4) | (senselT >> 8));
        Dest[tt++] = (unsigned char)(senselT & 0xff);

        Dest[tt++] = (unsigned char)((senselU >> 4) & 0xff);
        Dest[tt++] = (unsigned char)(((senselU & 0xF) << 4) | (senselV >> 8));
        Dest[tt++] = (unsigned char)(senselV & 0xff);

        Dest[tt++] = (unsigned char)((senselW >> 4) & 0xff);
        Dest[tt++] = (unsigned char)(((senselW & 0xF) << 4) | (senselX >> 8));
        Dest[tt++] = (unsigned char)(senselX & 0xff);

    }
    return std::move(Dest);
}

